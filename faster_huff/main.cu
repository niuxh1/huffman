#include <iostream>
#include <fstream>
#include <unordered_map>
#include "huffman.h"

using namespace std;

string file_path = R"(C:\Users\ASUS\Desktop\clion\faster_huff\)";

int main() {
  try {
    // 创建 huffman 对象

    huffman huff;

    // 构建字符频率映射
    unordered_map<char, int> freq_map;

    // 示例输入文件，计算字符频率
    string filename = file_path+"data_freq.txt";
    ifstream input_file(filename);
    if (!input_file) {
      throw runtime_error("Error: Could not open input file for frequency calculation!");
    }

    char ch;
    while (input_file.get(ch)) {
      freq_map[ch]++;
    }

    input_file.close();

    // 构建哈夫曼树和编码
    huff.build(freq_map);

    // 保存哈夫曼树到文件
    huff.save_tree(file_path+"hfm_tree.txt");

    // 读取哈夫曼树
    huff.read_tree(file_path+"hfm_tree.txt");

    auto start = chrono::high_resolution_clock::now();

    huff.encode(file_path+"data_source.txt", file_path+"data_encoded.txt");

    auto cpu_end = chrono::high_resolution_clock::now();

    // 编码文件 (CUDA版)
    huff.encode_cuda(file_path+"data_source.txt", file_path+"data_encoded_cuda.txt");

    auto gpu_end = chrono::high_resolution_clock::now();

    // 打印哈夫曼树结构到文件
    huff.print_tree(file_path+"print_tree.txt");

    cout << "Huffman compression completed!" << endl;

    cout<<"CPU time: "<<chrono::duration_cast<chrono::milliseconds>(cpu_end - start).count()<<"ms"<<endl;
    cout<<"GPU time: "<<chrono::duration_cast<chrono::milliseconds>(gpu_end - cpu_end).count()<<"ms"<<endl;

  } catch (const exception& e) {
    cerr << "Error: " << e.what() << endl;
    return 1;
  }
  return 0;
}
